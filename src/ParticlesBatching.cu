#include "ParticlesBatching.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include "Alloc.h"
#include "Parameters.h"
#include "PrecisionTypes.h"
#include "Grid.h"
#include "EMfield.h"
#include "InterpDensSpecies.h"

#define TPB 64


size_t queryFreeMemoryOnGPU(void)
{   
        size_t free_byte ;
        size_t total_byte ;

        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte );
        if ( hipSuccess != cuda_status ){
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }

    return (double)free_byte; //return amount of free memory on GPU in bytes
}

/* particle mover for GPU with batching */
/*
int mover_GPU_batch(struct particles* part, struct EMfield* field, struct grid* grd, struct parameters* param)
{
    // print species and subcycling
    std::cout << "***GPU MOVER with SUBCYCLYING "<< param->n_sub_cycles << " - species " << part->species_ID << " ***" << std::endl;

    // auxiliary variables
    FPpart dt_sub_cycling = (FPpart) param->dt/((double) part->n_sub_cycles);
    FPpart dto2 = .5*dt_sub_cycling, qomdt2 = part->qom*dto2/param->c;

    // allocate memory for variables on device

    FPpart *x_dev = NULL, *y_dev = NULL, *z_dev = NULL, *u_dev = NULL, *v_dev = NULL, *w_dev = NULL;
    FPinterp *q_dev = NULL;
    FPfield *XN_flat_dev = NULL, *YN_flat_dev = NULL, *ZN_flat_dev = NULL, *Ex_flat_dev = NULL, *Ey_flat_dev = NULL, *Ez_flat_dev = NULL, *Bxn_flat_dev = NULL, *Byn_flat_dev, *Bzn_flat_dev = NULL;

    size_t free_bytes = 0;
    int batch_number = 0;

    free_bytes = queryFreeMemoryOnGPU();

    size_t necessary_bytes = 0;

    const long int to = split_index + MAX_GPU_PARTICILES - 1 < part->npmax - 1 ? split_index + MAX_GPU_PARTICILES - 1 : part->npmax - 1;
    const int n_particles = to - split_index + 1;
    size_t batch_size = (to - split_index + 1) * sizeof(FPpart);


    hipMalloc(&x_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(x_dev, part->x, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&y_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(y_dev, part->y, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&z_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(z_dev, part->z, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&u_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(u_dev, part->u, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&v_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(v_dev, part->v, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&w_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(w_dev, part->w, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&q_dev, part->npmax * sizeof(FPinterp));
    hipMemcpy(q_dev, part->q, part->npmax * sizeof(FPinterp), hipMemcpyHostToDevice);  

    hipMalloc(&XN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(XN_flat_dev, grd->XN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&YN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(YN_flat_dev, grd->YN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&ZN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(ZN_flat_dev, grd->ZN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    
    hipMalloc(&Ex_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(Ex_flat_dev, field->Ex_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&Ey_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(Ey_flat_dev, field->Ey_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&Ez_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(Ez_flat_dev, field->Ez_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&Bxn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(Bxn_flat_dev, field->Bxn_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&Byn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(Byn_flat_dev, field->Byn_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&Bzn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(Bzn_flat_dev, field->Bzn_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);


    const long int to = split_index + MAX_GPU_PARTICILES - 1 < part->npmax - 1 ? split_index + MAX_GPU_PARTICILES - 1 : part->npmax - 1;

    const int n_particles = to - split_index + 1;
    size_t batch_size = (to - split_index + 1) * sizeof(FPpart);        

    FPpart *d_x, *d_y, *d_z, *d_u, *d_v, *d_w;
    hipMalloc(&d_x, batch_size);
    hipMalloc(&d_y, batch_size);
    hipMalloc(&d_z, batch_size);
    hipMalloc(&d_u, batch_size);
    hipMalloc(&d_v, batch_size);
    hipMalloc(&d_w, batch_size);

    //particles
    hipMemcpy(d_x, part->x+split_index, batch_size, hipMemcpyHostToDevice); 
    hipMemcpy(d_y, part->y+split_index, batch_size, hipMemcpyHostToDevice); 
    hipMemcpy(d_z, part->z+split_index, batch_size, hipMemcpyHostToDevice);
    hipMemcpy(d_u, part->u+split_index, batch_size, hipMemcpyHostToDevice); 
    hipMemcpy(d_v, part->v+split_index, batch_size, hipMemcpyHostToDevice); 
    hipMemcpy(d_w, part->w+split_index, batch_size, hipMemcpyHostToDevice); 

    // start subcycling
    for (int i_sub=0; i_sub <  part->n_sub_cycles; i_sub++){

        // Call GPU kernel
        single_particle_kernel<<<(part->npmax + TPB - 1)/TPB, TPB>>>(x_dev, y_dev, z_dev,u_dev, v_dev, w_dev, q_dev, XN_flat_dev, YN_flat_dev, ZN_flat_dev, grd->nxn, grd->nyn, grd->nzn, grd->xStart, grd->yStart, grd->zStart, grd->invdx, grd->invdy, grd->invdz, grd->Lx, grd->Ly, grd->Lz, grd->invVOL, Ex_flat_dev, Ey_flat_dev, Ez_flat_dev, Bxn_flat_dev, Byn_flat_dev, Bzn_flat_dev, param->PERIODICX, param->PERIODICY, param->PERIODICZ, dt_sub_cycling, dto2, qomdt2, part->NiterMover, part->nop);
        hipDeviceSynchronize();

    } // end of one particle


    // copy memory back to CPU (only the parts that have been modified inside the kernel)

    hipMemcpy(part->x, x_dev, part->npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
    hipMemcpy(part->y, y_dev, part->npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
    hipMemcpy(part->z, z_dev, part->npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
    hipMemcpy(part->u, u_dev, part->npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
    hipMemcpy(part->v, v_dev, part->npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
    hipMemcpy(part->w, w_dev, part->npmax * sizeof(FPpart), hipMemcpyDeviceToHost);
        
    hipMemcpy(field->Ex_flat, Ex_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Ey_flat, Ey_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Ez_flat, Ez_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Bxn_flat, Bxn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Byn_flat, Byn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Bzn_flat, Bzn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    
    // clean up
   
    hipFree(x_dev);
    hipFree(y_dev);
    hipFree(z_dev);
    hipFree(u_dev);
    hipFree(v_dev);
    hipFree(w_dev);

    hipFree(XN_flat_dev);
    hipFree(YN_flat_dev);
    hipFree(ZN_flat_dev);

    hipFree(Ex_flat_dev);
    hipFree(Ey_flat_dev);
    hipFree(Ez_flat_dev);
    hipFree(Bxn_flat_dev);
    hipFree(Byn_flat_dev);
    hipFree(Bzn_flat_dev);

    return(0);
}


void interpP2G_GPU_batch(struct particles* part, struct interpDensSpecies* ids, struct grid* grd)
{

    FPpart *x_dev = NULL, *y_dev = NULL, *z_dev = NULL, *u_dev = NULL, *v_dev = NULL, *w_dev = NULL;
    FPinterp * q_dev = NULL, *Jx_flat_dev = NULL, *Jy_flat_dev = NULL, *Jz_flat_dev = NULL, *rhon_flat_dev = NULL, *pxx_flat_dev = NULL, *pxy_flat_dev = NULL, *pxz_flat_dev = NULL, *pyy_flat_dev = NULL, *pyz_flat_dev = NULL, *pzz_flat_dev = NULL;
    FPfield *XN_flat_dev = NULL, *YN_flat_dev = NULL, *ZN_flat_dev = NULL;

    hipMalloc(&x_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(x_dev, part->x, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&y_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(y_dev, part->y, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&z_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(z_dev, part->z, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&u_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(u_dev, part->u, part->npmax* sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&v_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(v_dev, part->v, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&w_dev, part->npmax * sizeof(FPpart));
    hipMemcpy(w_dev, part->w, part->npmax * sizeof(FPpart), hipMemcpyHostToDevice);

    hipMalloc(&q_dev, part->npmax * sizeof(FPinterp));
    hipMemcpy(q_dev, part->q, part->npmax * sizeof(FPinterp), hipMemcpyHostToDevice);

    hipMalloc(&Jx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMemcpy(Jx_flat_dev, ids->Jx_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&Jy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMemcpy(Jy_flat_dev, ids->Jy_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&Jz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMemcpy(Jz_flat_dev, ids->Jz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&rhon_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMemcpy(rhon_flat_dev, ids->rhon_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&pxx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMemcpy(pxx_flat_dev, ids->pxx_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&pxy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMemcpy(pxy_flat_dev, ids->pxy_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&pxz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMemcpy(pxz_flat_dev, ids->pxz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&pyy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMemcpy(pyy_flat_dev, ids->pyy_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&pyz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMemcpy(pyz_flat_dev, ids->pyz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&pzz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMemcpy(pzz_flat_dev, ids->pzz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&XN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(XN_flat_dev, grd->XN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&YN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(YN_flat_dev, grd->YN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    hipMalloc(&ZN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMemcpy(ZN_flat_dev, grd->ZN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    interP2G_kernel<<<(part->npmax + TPB - 1)/TPB, TPB>>>(  x_dev, y_dev, z_dev, u_dev, v_dev, w_dev, q_dev, XN_flat_dev, YN_flat_dev, ZN_flat_dev, grd->nxn, grd->nyn, grd->nzn, grd->xStart, grd->yStart, grd->zStart, grd->invdx, grd->invdy, grd->invdz, grd->invVOL, Jx_flat_dev, Jy_flat_dev, Jz_flat_dev, rhon_flat_dev, pxx_flat_dev , pxy_flat_dev, pxz_flat_dev, pyy_flat_dev, pyz_flat_dev, pzz_flat_dev, part->nop);

    hipDeviceSynchronize();

    // copy memory back to CPU (only the parts that have been modified inside the kernel)

    hipMemcpy(ids->Jx_flat, Jx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->Jy_flat, Jy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->Jz_flat, Jz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->rhon_flat, rhon_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxx_flat, pxx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxy_flat, pxy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxz_flat, pxz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pyy_flat, pyy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pyz_flat, pyz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pzz_flat, pzz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    
    // clean up

    hipFree(x_dev);
    hipFree(y_dev);
    hipFree(z_dev);
    hipFree(u_dev);
    hipFree(v_dev);
    hipFree(w_dev);
    hipFree(q_dev);
    hipFree(XN_flat_dev);
    hipFree(YN_flat_dev);
    hipFree(ZN_flat_dev);
    hipFree(rhon_flat_dev);
    hipFree(pxx_flat_dev);
    hipFree(pxy_flat_dev);
    hipFree(pxz_flat_dev);
    hipFree(pyy_flat_dev);
    hipFree(pyz_flat_dev);
    hipFree(pzz_flat_dev);

}
+/