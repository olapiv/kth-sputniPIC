#include "hip/hip_runtime.h"
#include "ParticlesBatching.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include "Alloc.h"
#include "Parameters.h"
#include "Particles.h"
#include "PrecisionTypes.h"
#include "Grid.h"
#include "EMfield.h"
#include "InterpDensSpecies.h"

#define TPB 64


size_t queryFreeMemoryOnGPU(void)
{   
        size_t free_byte;
        size_t total_byte;

        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte );
        if ( hipSuccess != cuda_status ){
            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
            exit(1);
        }

    return free_byte * 0.8; // Assume 20% for safety
}

/* particle mover for GPU with batching */
int mover_GPU_batch(struct particles* part, struct EMfield* field, struct grid* grd, struct parameters* param)
{
    // print species and subcycling
    std::cout << "***GPU MOVER with SUBCYCLYING "<< param->n_sub_cycles << " - species " << part->species_ID << " ***" << std::endl;

    // auxiliary variables
    FPpart dt_sub_cycling = (FPpart) param->dt/((double) part->n_sub_cycles);
    FPpart dto2 = .5*dt_sub_cycling, qomdt2 = part->qom*dto2/param->c;

    // allocate memory for variables on device
    FPinterp *q_dev = NULL;
    FPfield *XN_flat_dev = NULL, *YN_flat_dev = NULL, *ZN_flat_dev = NULL, *Ex_flat_dev = NULL, *Ey_flat_dev = NULL, *Ez_flat_dev = NULL, *Bxn_flat_dev = NULL, *Byn_flat_dev, *Bzn_flat_dev = NULL;

    // Necesssary for all batches:
    hipMalloc(&q_dev, part->npmax * sizeof(FPinterp));
    hipMalloc(&XN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&YN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&ZN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Ex_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Ey_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Ez_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Bxn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Byn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&Bzn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));

    hipMemcpy(q_dev, part->q, part->npmax * sizeof(FPinterp), hipMemcpyHostToDevice);
    hipMemcpy(XN_flat_dev, grd->XN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(YN_flat_dev, grd->YN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(ZN_flat_dev, grd->ZN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Ex_flat_dev, field->Ex_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Ey_flat_dev, field->Ey_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Ez_flat_dev, field->Ez_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Bxn_flat_dev, field->Bxn_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Byn_flat_dev, field->Byn_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Bzn_flat_dev, field->Bzn_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
     
    // Particles to split up
    FPpart *x_dev = NULL, *y_dev = NULL, *z_dev = NULL, *u_dev = NULL, *v_dev = NULL, *w_dev = NULL;

    size_t free_bytes = queryFreeMemoryOnGPU();
    size_t total_necessary_bytes = 6 * part->npmax * sizeof(FPpart);
    int number_of_batches = static_cast<int>(ceil(total_necessary_bytes / free_bytes));
    size_t size_per_attribute_per_batch = free_bytes / 6;
    int max_num_particles_gpu = static_cast<int>(floor(((size_per_attribute_per_batch / 6) / sizeof(FPpart))));

    /* 
    const long int to = split_index + MAX_GPU_PARTICILES - 1 < part->npmax - 1 ? split_index + MAX_GPU_PARTICILES - 1 : part->npmax - 1;
    const int n_particles = to - split_index + 1;
    size_t batch_size = (to - split_index + 1) * sizeof(FPpart);
    */

    hipMalloc(&x_dev, size_per_attribute_per_batch);
    hipMalloc(&y_dev, size_per_attribute_per_batch);
    hipMalloc(&z_dev, size_per_attribute_per_batch);
    hipMalloc(&u_dev, size_per_attribute_per_batch);
    hipMalloc(&v_dev, size_per_attribute_per_batch);
    hipMalloc(&w_dev, size_per_attribute_per_batch);

    int split_index;

    for (int n_batch = 0; n_batch < number_of_batches; n_batch++) {

        split_index = n_batch * max_num_particles_gpu;

        hipMemcpy(x_dev, &(part->x[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice); 
        hipMemcpy(y_dev, &(part->y[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice); 
        hipMemcpy(z_dev, &(part->z[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice);
        hipMemcpy(u_dev, &(part->u[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice); 
        hipMemcpy(v_dev, &(part->v[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice); 
        hipMemcpy(w_dev, &(part->w[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice); 

        // start subcycling
        for (int i_sub=0; i_sub <  part->n_sub_cycles; i_sub++){

            // Call GPU kernel
            single_particle_kernel<<<(part->npmax + TPB - 1)/TPB, TPB>>>(
                x_dev, y_dev, z_dev, u_dev, v_dev, w_dev, q_dev, XN_flat_dev, YN_flat_dev, ZN_flat_dev, 
                grd->nxn, grd->nyn, grd->nzn, grd->xStart, grd->yStart, grd->zStart, 
                grd->invdx, grd->invdy, grd->invdz, grd->Lx, grd->Ly, grd->Lz, grd->invVOL, 
                Ex_flat_dev, Ey_flat_dev, Ez_flat_dev, Bxn_flat_dev, Byn_flat_dev, Bzn_flat_dev, 
                param->PERIODICX, param->PERIODICY, param->PERIODICZ, 
                dt_sub_cycling, dto2, qomdt2, 
                part->NiterMover, part->nop
            );
            hipDeviceSynchronize();

        } // end of one particle

        hipMemcpy(part->x, x_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);
        hipMemcpy(part->y, y_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);
        hipMemcpy(part->z, z_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);
        hipMemcpy(part->u, u_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);
        hipMemcpy(part->v, v_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);
        hipMemcpy(part->w, w_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);

    }

    hipFree(x_dev);
    hipFree(y_dev);
    hipFree(z_dev);
    hipFree(u_dev);
    hipFree(v_dev);
    hipFree(w_dev);

    // Copy memory back to CPU (only the parts that have been modified inside the kernel)
        
    hipMemcpy(field->Ex_flat, Ex_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Ey_flat, Ey_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Ez_flat, Ez_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Bxn_flat, Bxn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Byn_flat, Byn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    hipMemcpy(field->Bzn_flat, Bzn_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyDeviceToHost);
    
    // Clean up
    hipFree(XN_flat_dev);
    hipFree(YN_flat_dev);
    hipFree(ZN_flat_dev);

    hipFree(Ex_flat_dev);
    hipFree(Ey_flat_dev);
    hipFree(Ez_flat_dev);
    hipFree(Bxn_flat_dev);
    hipFree(Byn_flat_dev);
    hipFree(Bzn_flat_dev);

    return(0);
}


void interpP2G_GPU_batch(struct particles* part, struct interpDensSpecies* ids, struct grid* grd)
{

    // Necesssary for all batches:
    FPinterp * q_dev = NULL, *Jx_flat_dev = NULL, *Jy_flat_dev = NULL, *Jz_flat_dev = NULL, *rhon_flat_dev = NULL, *pxx_flat_dev = NULL, *pxy_flat_dev = NULL, *pxz_flat_dev = NULL, *pyy_flat_dev = NULL, *pyz_flat_dev = NULL, *pzz_flat_dev = NULL;
    FPfield *XN_flat_dev = NULL, *YN_flat_dev = NULL, *ZN_flat_dev = NULL;

    hipMalloc(&q_dev, part->npmax * sizeof(FPinterp));
    hipMalloc(&Jx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&Jy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&Jz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&rhon_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pxx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pxy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pxz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pyy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pyz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&pzz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp));
    hipMalloc(&XN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&YN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));
    hipMalloc(&ZN_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield));

    hipMemcpy(q_dev, part->q, part->npmax * sizeof(FPinterp), hipMemcpyHostToDevice);
    hipMemcpy(Jx_flat_dev, ids->Jx_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Jy_flat_dev, ids->Jy_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(Jz_flat_dev, ids->Jz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(rhon_flat_dev, ids->rhon_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pxx_flat_dev, ids->pxx_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pxy_flat_dev, ids->pxy_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pxz_flat_dev, ids->pxz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pyy_flat_dev, ids->pyy_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pyz_flat_dev, ids->pyz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(pzz_flat_dev, ids->pzz_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(XN_flat_dev, grd->XN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(YN_flat_dev, grd->YN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);
    hipMemcpy(ZN_flat_dev, grd->ZN_flat, grd->nxn * grd->nyn * grd->nzn * sizeof(FPfield), hipMemcpyHostToDevice);

    // Particles to split up
    FPpart *x_dev = NULL, *y_dev = NULL, *z_dev = NULL, *u_dev = NULL, *v_dev = NULL, *w_dev = NULL;

    size_t free_bytes = queryFreeMemoryOnGPU();
    size_t total_necessary_bytes = 6 * part->npmax * sizeof(FPpart);
    int number_of_batches = static_cast<int>(ceil(total_necessary_bytes / free_bytes));
    size_t size_per_attribute_per_batch = free_bytes / 6;
    int max_num_particles_gpu = static_cast<int>(floor(((size_per_attribute_per_batch / 6) / sizeof(FPpart))));

    /* 
    const long int to = split_index + MAX_GPU_PARTICILES - 1 < part->npmax - 1 ? split_index + MAX_GPU_PARTICILES - 1 : part->npmax - 1;
    const int n_particles = to - split_index + 1;
    size_t batch_size = (to - split_index + 1) * sizeof(FPpart);
    */

    hipMalloc(&x_dev, size_per_attribute_per_batch);
    hipMalloc(&y_dev, size_per_attribute_per_batch);
    hipMalloc(&z_dev, size_per_attribute_per_batch);
    hipMalloc(&u_dev, size_per_attribute_per_batch);
    hipMalloc(&v_dev, size_per_attribute_per_batch);
    hipMalloc(&w_dev, size_per_attribute_per_batch);

    int split_index;
    for (int n_batch = 0; n_batch < number_of_batches; n_batch++) {

        split_index = n_batch * max_num_particles_gpu;

        hipMemcpy(x_dev, &(part->x[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice); 
        hipMemcpy(y_dev, &(part->y[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice); 
        hipMemcpy(z_dev, &(part->z[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice);
        hipMemcpy(u_dev, &(part->u[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice); 
        hipMemcpy(v_dev, &(part->v[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice); 
        hipMemcpy(w_dev, &(part->w[split_index]), size_per_attribute_per_batch, hipMemcpyHostToDevice);

        interP2G_kernel<<<(part->npmax + TPB - 1)/TPB, TPB>>>(
            x_dev, y_dev, z_dev, u_dev, v_dev, w_dev, q_dev, 
            XN_flat_dev, YN_flat_dev, ZN_flat_dev, 
            grd->nxn, grd->nyn, grd->nzn, grd->xStart, grd->yStart, grd->zStart, 
            grd->invdx, grd->invdy, grd->invdz, grd->invVOL, 
            Jx_flat_dev, Jy_flat_dev, Jz_flat_dev, 
            rhon_flat_dev, pxx_flat_dev , pxy_flat_dev, pxz_flat_dev, pyy_flat_dev, pyz_flat_dev, pzz_flat_dev, 
            part->nop
        );
        hipDeviceSynchronize();

        hipMemcpy(part->x, x_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);
        hipMemcpy(part->y, y_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);
        hipMemcpy(part->z, z_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);
        hipMemcpy(part->u, u_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);
        hipMemcpy(part->v, v_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);
        hipMemcpy(part->w, w_dev, size_per_attribute_per_batch, hipMemcpyDeviceToHost);

    }

    // copy memory back to CPU (only the parts that have been modified inside the kernel)

    hipMemcpy(ids->Jx_flat, Jx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->Jy_flat, Jy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->Jz_flat, Jz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->rhon_flat, rhon_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxx_flat, pxx_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxy_flat, pxy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pxz_flat, pxz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pyy_flat, pyy_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pyz_flat, pyz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    hipMemcpy(ids->pzz_flat, pzz_flat_dev, grd->nxn * grd->nyn * grd->nzn * sizeof(FPinterp), hipMemcpyDeviceToHost);
    
    // clean up

    hipFree(x_dev);
    hipFree(y_dev);
    hipFree(z_dev);
    hipFree(u_dev);
    hipFree(v_dev);
    hipFree(w_dev);
    hipFree(q_dev);
    hipFree(XN_flat_dev);
    hipFree(YN_flat_dev);
    hipFree(ZN_flat_dev);
    hipFree(rhon_flat_dev);
    hipFree(pxx_flat_dev);
    hipFree(pxy_flat_dev);
    hipFree(pxz_flat_dev);
    hipFree(pyy_flat_dev);
    hipFree(pyz_flat_dev);
    hipFree(pzz_flat_dev);

}
